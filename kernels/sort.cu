#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "sort.cuh"


__global__ void mergePassKernel(int *input, int *output, int width, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int start = tid * 2 * width;
    
    if (start >= size) return;

    int mid = min(start + width, size);
    int end = min(start + 2 * width, size);

    int i = start, j = mid, k = start;

    while (i < mid && j < end) {
        output[k++] = (input[i] <= input[j]) ? input[i++] : input[j++];
    }
    while (i < mid) output[k++] = input[i++];
    while (j < end) output[k++] = input[j++];
}

void gpuMergeSort(int *data, int size) {
    int *d_data1, *d_data2;
    hipMalloc(&d_data1, size * sizeof(int));
    hipMalloc(&d_data2, size * sizeof(int));

    hipMemcpy(d_data1, data, size * sizeof(int), hipMemcpyHostToDevice);

    int *in = d_data1;
    int *out = d_data2;
    int width = 1;

    while (width < size) {
        int blocks = (size + 2 * width - 1) / (2 * width);
        mergePassKernel<<<blocks, 256>>>(in, out, width, size);
        hipDeviceSynchronize();

        int *temp = in;
        in = out;
        out = temp;

        width *= 2;
    }

    hipMemcpy(data, in, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data1);
    hipFree(d_data2);
}