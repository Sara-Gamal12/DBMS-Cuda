#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include "agg.cuh"
#include "get.cuh"

__global__ void max_kernel(char *input_data, int row_size, int acc_col_size, double *max_element, int n)
{
    // n=num of rows in given chunk
    extern __shared__ double partial_max[];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    // load 1st element data into shared memory
    if (start + t < n)
    {
        char *data_ptr = &input_data[(start + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_max[t] = INT_MIN;
        else
            memcpy(&partial_max[t], data_ptr, sizeof(double));
    }
    else
        partial_max[t] = INT_MIN;
    // load 2nd element data into shared memory
    if (start + blockDim.x + t < n)
    {
        char *data_ptr = &input_data[(start + blockDim.x + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_max[blockDim.x + t] = INT_MIN;
        else
            memcpy(&partial_max[blockDim.x + t], data_ptr, sizeof(double));
    }
    else
        partial_max[blockDim.x + t] = INT_MIN;

    // loop to reduce the data in shared memory
    // each thread will be responsible for 2 elements
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (t < stride)
            partial_max[t] = max(partial_max[t + stride], partial_max[t]);
    }
    __syncthreads();
    // write the result for this block to global memory
    if (t == 0)
    {
        max_element[blockIdx.x] = partial_max[0];
    }
}

__global__ void min_kernel(char *input_data, int row_size, int acc_col_size, double *min_element, int n)
{
    extern __shared__ double partial_min[];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    // load 1st element data into shared memory
    if (start + t < n)
    {
        char *data_ptr = &input_data[(start + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_min[t] = INT_MAX;
        else
            memcpy(&partial_min[t], data_ptr, sizeof(double));
    }
    else
        partial_min[t] = INT_MAX;
    // load 2nd element data into shared memory
    if (start + blockDim.x + t < n)
    {
        char *data_ptr = &input_data[(start + blockDim.x + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_min[blockDim.x + t] = INT_MAX;
        else
            memcpy(&partial_min[blockDim.x + t], data_ptr, sizeof(double));
    }
    else
        partial_min[blockDim.x + t] = INT_MAX;
    // loop to reduce the data in shared memory
    // each thread will be responsible for 2 elements
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (t < stride)
            if (t + stride < 2 * blockDim.x)
                partial_min[t] = min(partial_min[t + stride], partial_min[t]);
    }
    __syncthreads();
    // write the result for this block to global memory
    if (t == 0)
    {
        min_element[blockIdx.x] = partial_min[0];
    }
}

__global__ void sum_kernel(char *input_data, int row_size, int acc_col_size, double *sum_element, int n, int *num_rows)
{
    extern __shared__ double partial_sum[];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    // load 1st element data into shared memory
    if (start + t < n)
    {
        char *data_ptr = &input_data[(start + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_sum[t] = 0;
        else
        {
            atomicAdd(num_rows, 1);
            memcpy(&partial_sum[t], data_ptr, sizeof(double));
        }
    }
    else
        partial_sum[t] = 0;
    // load 2nd element data into shared memory
    if (start + blockDim.x + t < n)
    {

        char *data_ptr = &input_data[(start + blockDim.x + t) * row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") == 0)
            partial_sum[blockDim.x + t] = 0;
        else
        {
            atomicAdd(num_rows, 1);
            memcpy(&partial_sum[blockDim.x + t], data_ptr, sizeof(double));
        }
    }
    else
        partial_sum[blockDim.x + t] = 0;
    // loop to reduce the data in shared memory
    // each thread will be responsible for 2 elements
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (t < stride)
            if (t + stride < 2 * blockDim.x)
                partial_sum[t] = partial_sum[t + stride] + partial_sum[t];
    }
    __syncthreads();
    // write the result for this block to global memory
    if (t == 0)
    {
        sum_element[blockIdx.x] = partial_sum[0];
    }
}

__global__ void count_kernel(char *input_data, int row_size, int acc_col_size, int n, int *num_rows)
{
    unsigned int t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t < n)
    {
        char *data_ptr = &input_data[(t)*row_size + acc_col_size];
        if (device_strcmp(data_ptr, "NULL") != 0)
        {
            atomicAdd(&num_rows[blockIdx.x], 1);
        }
    }
}

__host__ double call_agg_kernel(char *input_data, int row_size, int acc_col_size, char *op, int n)
{
    char *d_input_data;
    double *d_output_data;
    double *h_output_data;
    int *d_num_rows;

    // Allocate device memory
    hipMalloc((void **)&d_input_data, n * row_size * sizeof(char));
    hipMemcpy(d_input_data, input_data, n * row_size * sizeof(char), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + (blockSize * 2) - 1) / (blockSize * 2);
    h_output_data = (double *)malloc(numBlocks * sizeof(double));
    hipMalloc((void **)&d_output_data, numBlocks * sizeof(double));

    size_t shared = 2 * blockSize * sizeof(double);
    ;
    if (strcmp(op, "max") == 0)
        max_kernel<<<numBlocks, blockSize, shared>>>(d_input_data, row_size, acc_col_size, d_output_data, n);
    else if (strcmp(op, "min") == 0)
        min_kernel<<<numBlocks, blockSize, shared>>>(d_input_data, row_size, acc_col_size, d_output_data, n);
    else if (strcmp(op, "sum") == 0 || strcmp(op, "avg") == 0)

    {
        hipMalloc((void **)&d_num_rows, sizeof(int));
        hipMemset(d_num_rows, 0, sizeof(int));
        sum_kernel<<<numBlocks, blockSize, shared>>>(d_input_data, row_size, acc_col_size, d_output_data, n, d_num_rows);
    }
    else if (strcmp(op, "count") == 0)
    {
        numBlocks = (n + (blockSize) - 1) / (blockSize);
        hipMalloc((void **)&d_num_rows, numBlocks*sizeof(int));
        hipMemset(d_num_rows, 0, numBlocks*sizeof(int));
        count_kernel<<<numBlocks, blockSize>>>(d_input_data, row_size, acc_col_size, n, d_num_rows);
    }

    // copy back the data
    hipMemcpy(h_output_data, d_output_data, numBlocks * sizeof(double), hipMemcpyDeviceToHost);

    // 1. Check for *launch* errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Kernel launch failed: %s \n", hipGetErrorString(err));
    }

    // 2. Check for *asynchronous* errors (e.g., during execution)
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        printf("Kernel launch failed: %s \n", hipGetErrorString(err));
    }

    double result;
    if (strcmp(op, "max") == 0)
    {
        result = INT_MIN;
        for (int i = 0; i < numBlocks; i++)
            result = max(result, h_output_data[i]);
    }
    else if (strcmp(op, "min") == 0)
    {
        result = INT_MAX;
        for (int i = 0; i < numBlocks; i++)
            result = min(result, h_output_data[i]);
    }
    else if (strcmp(op, "sum") == 0 || strcmp(op, "avg") == 0)
    {
        result = 0;
        for (int i = 0; i < numBlocks; i++)
            result += h_output_data[i];

        if (strcmp(op, "avg") == 0)
        {
            int num_rows = 0;
            hipMemcpy(&num_rows, d_num_rows, sizeof(int), hipMemcpyDeviceToHost);
            result /= num_rows;
        }
    }
    else if (strcmp(op, "count") == 0)
    {
        int * num_rows=new int[numBlocks];
        hipMemcpy(num_rows, d_num_rows, numBlocks*sizeof(int), hipMemcpyDeviceToHost);
        result = 0;
        for (int i = 0; i < numBlocks; i++)
            result += num_rows[i];
    }

    // Cleanup
    hipFree(d_input_data);
    hipFree(d_output_data);
    hipFree(d_num_rows);
    free(h_output_data);

    return result;
}
