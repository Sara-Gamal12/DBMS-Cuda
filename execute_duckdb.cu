#include "hip/hip_runtime.h"
#include "duckdb.hpp"
#include "duckdb/main/client_context.hpp"
#include "duckdb/parser/parser.hpp"
#include "duckdb/planner/planner.hpp"
#include "duckdb/optimizer/optimizer.hpp"
#include "duckdb/planner/logical_operator.hpp"
#include "duckdb/planner/operator/logical_get.hpp"
#include "duckdb/planner/operator/logical_comparison_join.hpp"
#include "duckdb/catalog/catalog_entry/table_catalog_entry.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/parser/expression/constant_expression.hpp"
#include "duckdb/common/enums/expression_type.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/planner/filter/constant_filter.hpp"
#include "duckdb/planner/filter/null_filter.hpp"
#include "duckdb/planner/filter/conjunction_filter.hpp"
#include "duckdb/planner/operator/logical_order.hpp"
#include "duckdb/execution/executor.hpp"

#include "duckdb/common/common.hpp"
#include "duckdb/common/enums/pending_execution_result.hpp"
#include "duckdb/common/mutex.hpp"
// #include "dukdb/common/pair.hpp"
#include "duckdb/common/reference_map.hpp"
#include "duckdb/main/query_result.hpp"
#include "duckdb/execution/task_error_manager.hpp"
#include "duckdb/execution/progress_data.hpp"
#include "duckdb/parallel/pipeline.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "./kernels/agg.cuh"
#include "./kernels/get.cuh"
#include "./kernels/project.cuh"

#include "./utilities/schema_utilities.hpp"
#include "./utilities/filter_utilities.hpp"

#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

using namespace std;
using namespace duckdb;

long RAM = 4 * pow(1024, 3); // 4GB
Schema schema;

struct PlanNode
{
    std::string name;
    std::vector<std::string> details; // e.g., filters, expressions
    std::vector<std::shared_ptr<PlanNode>> children;
};

struct return_node_type
{
    std::vector<char> data;
    std::vector<ColumnInfo> data_schema;
    int num_row;
};

return_node_type post_order_traverse_and_launch_kernel(std::shared_ptr<PlanNode> node)
{
    if (!node)
        return {};

    // 1. Traverse children first (post-order)
    std::vector<return_node_type> child_results;
    for (auto &child : node->children)
    {
        return_node_type data = post_order_traverse_and_launch_kernel(child);
        child_results.push_back(data);
    }

    // 2. Process current node (e.g., launch kernel)
    std::cout << "Launching kernel for operator: " << node->name << std::endl;

    // You can decide which CUDA kernel to call based on node->name
    if (node->name == "GET")
    {

        string table_name = node->details[0];
        int row_size;
        std::vector<char> chunk = read_csv_chunk(table_name, 0.5 * RAM, row_size);
        if (node->details.size() > 1)
        {

            Condition *conditions = new Condition[node->details.size() - 1];
            std::vector<string> expr;
            for (size_t i = 1; i < node->details.size(); ++i)
            {
                expr.push_back(node->details[i]);
                if (i != node->details.size() - 1)
                    expr.push_back("and");
            }
            int *acc_sums = new int[(schema[table_name].second.size())];
            std::vector<Token> tokens = tokenize(expr);
            std::vector<std::string> postfix = infix_to_postfix(tokens);
            std::vector<ConditionToken> condition_tokens = parse_postfix(postfix, schema[table_name].second, acc_sums);

            int n = chunk.size() / row_size;
            int output_counter = 0;
            char *data = call_get_kernel(chunk.data(), row_size, acc_sums, condition_tokens, condition_tokens.size(), n, output_counter, schema[table_name].second.size());
            return_node_type return_data;
            return_data.data = std::vector<char>(data, data + output_counter * row_size);
            return_data.num_row = output_counter;
            return_data.data_schema = schema[table_name].second;
            return return_data;
        }
        else
        {
            return_node_type return_data;
            return_data.data = chunk;
            return_data.num_row = chunk.size() / row_size;
            return_data.data_schema = schema[table_name].second;
            return return_data;
        }

        // launch_get_kernel();  // Your kernel logic here
    }

    else if (node->name == "FILTER")
    {
        int row_size = child_results[0].data.size() / child_results[0].num_row;
        Condition *conditions = new Condition[node->details.size() - 1];
        std::string expr = node->details[0];
        std::string to_remove = "::TIMESTAMP";
        size_t pos = expr.find(to_remove);
        if (pos != std::string::npos)
        {
            expr.erase(pos, to_remove.length());
        }

        expr = replace_operatirs(expr);
        int *acc_sums = new int[child_results[0].data_schema.size()];
        std::vector<std::string> vector_expr = tokenizeExpression(expr);

        std::vector<Token> tokens = tokenize(vector_expr);
        std::vector<std::string> postfix = infix_to_postfix(tokens);
        std::vector<ConditionToken> condition_tokens = parse_postfix(postfix, child_results[0].data_schema, acc_sums);

        int output_counter = 0;
        char *data = call_get_kernel(child_results[0].data.data(), row_size, acc_sums, condition_tokens, condition_tokens.size(), child_results[0].num_row, output_counter, child_results[0].data_schema.size());
        return_node_type return_data;
        return_data.data = std::vector<char>(data, data + output_counter * row_size);
        return_data.num_row = output_counter;
        return_data.data_schema = child_results[0].data_schema;
        return return_data;
    }
    else if (node->name == "JOIN")
    {
        // launch_join_kernel(); // Your kernel logic here
    }
    else if (node->name == "ORDERBY")
    {
        // launch_order_kernel(); // Your kernel logic here
    }
    else if (node->name == "AGGREGATE")
    {
        char *op;
        string col_name = node->details[0].substr(node->details[0].find("(") + 1, node->details[0].find(")") - node->details[0].find("(") - 1);
        int acc_col_size;
        int row_size = 0;
        int index;
        for (int i = 0; i < child_results[0].data_schema.size(); i++)
        {
            if (child_results[0].data_schema[i].name == col_name)
            {
                index = i;
                acc_col_size = child_results[0].data_schema[i].acc_col_size;
            }
            row_size += child_results[0].data_schema[i].size_in_bytes;
        }

        if (node->details[0].find("max") != std::string::npos)
        {
            op = "max";
        }
        else if (node->details[0].find("min") != std::string::npos)
        {
            op = "min";
        }
        else if (node->details[0].find("avg") != std::string::npos)
        {
            op = "avg";
        }
        else if (node->details[0].find("sum") != std::string::npos)
        {
            op = "sum";
        }
        else if (node->details[0].find("count") != std::string::npos)
        {
            op = "count";
        }

        double result = call_agg_kernel(child_results[0].data.data(), row_size, acc_col_size, op, child_results[0].num_row);
        const char *result_str = reinterpret_cast<const char *>(&result);
        return_node_type return_data;
        return_data.data = std::vector<char>(result_str, result_str + sizeof(result));
        return_data.num_row = 1;
        ColumnInfo col_info;
        col_info.type = child_results[0].data_schema[index].type;
        col_info.size_in_bytes = sizeof(result);
        col_info.acc_col_size = 0;
        col_info.name = node->details[0];
        return_data.data_schema.push_back(col_info);
        return return_data;
    }
    else if (node->name == "PROJECTION")
    {
        int *acc_sums = new int[node->details.size()];
        int *col_index = new int[node->details.size()];
        int *sizes = new int[node->details.size()];
        int new_row_size = 0;
        std::vector<ColumnInfo> child_schema = child_results[0].data_schema;
        std::vector<ColumnInfo> new_schema;

        for (int i = 0; i < node->details.size(); i++)
        {
            std::string col_name = node->details[i];
            for (int j = 0; j < child_results[0].data_schema.size(); j++)
            {
                if (child_results[0].data_schema[j].name == col_name)
                {
                    new_schema.push_back(child_results[0].data_schema[j]);
                    col_index[i] = j;
                    acc_sums[i] = child_results[0].data_schema[j].acc_col_size;
                    sizes[i] = child_results[0].data_schema[j].size_in_bytes;
                    new_row_size += child_results[0].data_schema[j].size_in_bytes;
                }
            }
        }

        int row_size = child_results[0].data.size() / child_results[0].num_row;

        char *data = call_project_kernel(child_results[0].data.data(), new_row_size, row_size, col_index, acc_sums, child_results[0].num_row, node->details.size(), sizes);
        return_node_type return_data;
        return_data.data = std::vector<char>(data, data + child_results[0].num_row * new_row_size);
        return_data.num_row = child_results[0].num_row;
        return_data.data_schema = new_schema;

        return return_data;
    }
    else
    {
        std::cout << "No matching kernel for: " << node->name << std::endl;
    }
}

std::shared_ptr<PlanNode> build_plan_tree(LogicalOperator *op)
{
    if (!op)
        return nullptr;

    auto node = std::make_shared<PlanNode>();
    node->name = LogicalOperatorToString(op->type);

    // Handle LogicalGet
    if (auto get_op = dynamic_cast<duckdb::LogicalGet *>(op))
    {
        if (get_op->GetTable())
        {
            node->details.push_back(get_op->GetTable()->name);
        }
        for (auto &[col_idx, filter] : get_op->table_filters.filters)
        {
            std::ostringstream oss;
            oss << get_op->names[col_idx] << " ";
            switch (filter->filter_type)
            {
            case duckdb::TableFilterType::CONSTANT_COMPARISON:
            {
                auto &f = static_cast<duckdb::ConstantFilter &>(*filter);
                std::string constant_str = f.constant.ToString();
                if (f.constant.type().id() == duckdb::LogicalTypeId::VARCHAR &&
                    constant_str.front() != '\'' && constant_str.back() != '\'')
                {
                    constant_str = "'" + constant_str + "'";
                }
                oss << ExpressionTypeToString(f.comparison_type) << " " << constant_str;
                break;
            }
            case duckdb::TableFilterType::IS_NULL:
                oss << "IS NULL";
                break;
            case duckdb::TableFilterType::IS_NOT_NULL:
                oss << "IS NOT NULL";
                break;
            case duckdb::TableFilterType::CONJUNCTION_AND:
                oss << "AND";
                break;
            case duckdb::TableFilterType::CONJUNCTION_OR:
                oss << "OR";
                break;
            default:
                oss << "UNKNOWN";
                break;
            }
            node->details.push_back(oss.str());
        }
    }

    // Handle LogicalComparisonJoin
    if (op->type == LogicalOperatorType::LOGICAL_COMPARISON_JOIN)
    {
        auto &join_op = static_cast<duckdb::LogicalComparisonJoin &>(*op);
        for (auto &condition : join_op.conditions)
        {
            node->details.push_back("Join: " + condition.left->ToString() + " " + ExpressionTypeToString(condition.comparison) + " " + condition.right->ToString());
        }
    }

    // Handle LogicalOrder
    if (op->type == LogicalOperatorType::LOGICAL_ORDER_BY)
    {
        auto &order_by_op = static_cast<duckdb::LogicalOrder &>(*op);
        for (auto &order : order_by_op.orders)
        {
            std::string order_str = "Order By: " + order.expression->ToString();
            order_str += (order.type == duckdb::OrderType::ASCENDING) ? " ASC" : " DESC";
            node->details.push_back(order_str);
        }
    }

    for (auto &expr : op->expressions)
    {
        node->details.push_back(expr->ToString());
    }

    // Recurse on children
    for (auto &child : op->children)
    {
        node->children.push_back(build_plan_tree(child.get()));
    }

    return node;
}

void print_tree(std::shared_ptr<PlanNode> node, int indent = 0)
{
    if (!node)
        return;
    std::cout << std::string(indent, ' ') << "- " << node->name << std::endl;
    for (const auto &detail : node->details)
    {
        std::cout << std::string(indent + 2, ' ') << "* " << detail << std::endl;
    }
    for (const auto &child : node->children)
    {
        print_tree(child, indent + 4);
    }
}

int main(int argc, char *argv[])
{
    // DuckDB
    using namespace duckdb;
    DuckDB db(nullptr);
    Connection con(db);
    ClientContext &context = *con.context;

    while (true)
    {
        cout << "\nEnter SQL query (or type 'exit' to quit): ";
        string query;
        getline(cin, query);

        if (query == "exit" || query == "quit")
        {
            cout << "Exiting CLI.\n";
            break;
        }
        get_schema(schema);
        create_tables_from_schema(con, schema);
        // string query = "select * from course  where id=Sid ;";
        // string query = "select sum(age) from student;";

        Parser parser;
        parser.ParseQuery(query);
        auto statements = std::move(parser.statements);
        // Start a transaction
        con.BeginTransaction(); // Start transaction using Connection

        // Create a planner and plan the query
        Planner planner(context);
        planner.CreatePlan(std::move(statements[0]));

        // Now you can proceed with further processing or optimization
        cout << "Planning successful!" << endl;
        cout << "Unoptimized Logical Plan:\n"
            << planner.plan->ToString() << endl;

        Optimizer optimizer(*planner.binder, context);
        auto logical_plan = optimizer.Optimize(std::move(planner.plan));
        cout << "Optimized Logical Plan:\n";
        cout << logical_plan->ToString() << endl;

        auto tree_root = build_plan_tree(logical_plan.get());
        print_tree(tree_root);

        // Traverse the plan tree and launch kernels
        return_node_type data_out = post_order_traverse_and_launch_kernel(tree_root);

        print_chunk(data_out.data, data_out.data_schema);
        // Commit the transaction after planning
        con.Commit(); // Commit transaction using Connection
    }
}
