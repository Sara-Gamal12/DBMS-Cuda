#include "hip/hip_runtime.h"
#include "duckdb.hpp"
#include "duckdb/main/client_context.hpp"
#include "duckdb/parser/parser.hpp"
#include "duckdb/planner/planner.hpp"
#include "duckdb/optimizer/optimizer.hpp"
#include "duckdb/planner/logical_operator.hpp"
#include "duckdb/planner/operator/logical_get.hpp"
#include "duckdb/planner/operator/logical_comparison_join.hpp"
#include "duckdb/catalog/catalog_entry/table_catalog_entry.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/parser/expression/constant_expression.hpp"
#include "duckdb/common/enums/expression_type.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/planner/table_filter.hpp"
#include "duckdb/planner/filter/constant_filter.hpp"
#include "duckdb/planner/filter/null_filter.hpp"
#include "duckdb/planner/filter/conjunction_filter.hpp"
#include "duckdb/planner/operator/logical_order.hpp"
#include "duckdb/execution/executor.hpp"

#include "duckdb/common/common.hpp"
#include "duckdb/common/enums/pending_execution_result.hpp"
#include "duckdb/common/mutex.hpp"
// #include "dukdb/common/pair.hpp"
#include "duckdb/common/reference_map.hpp"
#include "duckdb/main/query_result.hpp"
#include "duckdb/execution/task_error_manager.hpp"
#include "duckdb/execution/progress_data.hpp"
#include "duckdb/parallel/pipeline.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "./kernels/agg.cuh"
#include "./kernels/get.cuh"
#include "./kernels/project.cuh"
#include "./kernels/sort.cuh"

#include "./utilities/schema_utilities.hpp"
#include "./utilities/filter_utilities.hpp"

#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <regex>
using namespace std;
using namespace duckdb;

long RAM = 4 * pow(1024, 3); // 4GB
Schema schema;

struct PlanNode
{
    std::string name;
    std::vector<std::string> details; // e.g., filters, expressions
    std::vector<std::shared_ptr<PlanNode>> children;
};

struct return_node_type
{
    std::vector<char> data;
    std::vector<ColumnInfo> data_schema;
    int num_row;
};

return_node_type post_order_traverse_and_launch_kernel(std::shared_ptr<PlanNode> node)
{
    if (!node)
        return {};

    // 1. Traverse children first (post-order)
    std::vector<return_node_type> child_results;
    for (auto &child : node->children)
    {
        return_node_type data = post_order_traverse_and_launch_kernel(child);
        child_results.push_back(data);
    }

    // 2. Process current node (e.g., launch kernel)
    std::cout << "Launching kernel for operator: " << node->name << std::endl;

    if (child_results.size() != 0 && child_results[0].num_row == 0)
    {
        return child_results[0];
    }
    // You can decide which CUDA kernel to call based on node->name
    if (node->name == "GET")
    {

        string table_name = node->details[0];
        int row_size;
        std::vector<char> chunk = read_csv_chunk(table_name, 0.5 * RAM, row_size);
        if (node->details.size() > 1)
        {

            Condition *conditions = new Condition[node->details.size() - 1];
            std::vector<string> expr;
            for (size_t i = 1; i < node->details.size(); ++i)
            {
                expr.push_back(node->details[i]);
                if (i != node->details.size() - 1)
                    expr.push_back("and");
            }
            int *acc_sums = new int[(schema[table_name].second.size())];
            std::vector<Token> tokens = tokenize(expr);
            std::vector<std::string> postfix = infix_to_postfix(tokens);
            std::vector<ConditionToken> condition_tokens = parse_postfix(postfix, schema[table_name].second, acc_sums);

            int n = chunk.size() / row_size;
            int output_counter = 0;
            char *data = call_get_kernel(chunk.data(), row_size, acc_sums, condition_tokens, condition_tokens.size(), n, output_counter, schema[table_name].second.size());
            return_node_type return_data;
            return_data.data = std::vector<char>(data, data + output_counter * row_size);
            return_data.num_row = output_counter;
            return_data.data_schema = schema[table_name].second;
            return return_data;
        }
        else
        {
            return_node_type return_data;
            return_data.data = chunk;
            return_data.num_row = chunk.size() / row_size;
            return_data.data_schema = schema[table_name].second;
            return return_data;
        }

        // launch_get_kernel();  // Your kernel logic here
    }
    else if (node->name == "FILTER")
    {
        int row_size = child_results[0].data.size() / child_results[0].num_row;
        std::string expr = "";
        for (size_t i = 0; i < node->details.size(); ++i)
        {
            expr += node->details[i];
            if (i != node->details.size() - 1)
                expr += " and ";
        }
        std::string to_remove = "::TIMESTAMP";
        size_t pos;
        while ((pos = expr.find(to_remove)) != std::string::npos)
        {
            expr.erase(pos, to_remove.length());
        }
        expr = replace_operatirs(expr);
        int *acc_sums = new int[child_results[0].data_schema.size()];
        std::vector<std::string> vector_expr = tokenizeExpression(expr);

        std::vector<Token> tokens = tokenize(vector_expr);
        std::vector<std::string> postfix = infix_to_postfix(tokens);
        std::vector<ConditionToken> condition_tokens = parse_postfix(postfix, child_results[0].data_schema, acc_sums);

        int output_counter = 0;
        char *data = call_get_kernel(child_results[0].data.data(), row_size, acc_sums, condition_tokens, condition_tokens.size(), child_results[0].num_row, output_counter, child_results[0].data_schema.size());
        return_node_type return_data;
        return_data.data = std::vector<char>(data, data + output_counter * row_size);
        return_data.num_row = output_counter;
        return_data.data_schema = child_results[0].data_schema;
        return return_data;
    }
    else if (node->name == "JOIN")
    {
        // launch_join_kernel(); // Your kernel logic here
    }
    else if (node->name == "ORDER_BY")
    {
        size_t lastDot = node->details[0].rfind('.');
        size_t lastSpace = node->details[0].rfind(' ');

        string col_name = node->details[0].substr(lastDot + 1, lastSpace - lastDot - 1);
        string oredr_method = node->details[0].substr(lastSpace + 1);

        std::cout << "col_name: " << col_name << std::endl;
        std::cout << "oredr_method: " << oredr_method << std::endl;
        int row_size = child_results[0].data.size() / child_results[0].num_row;
        int i = 0;
        while (child_results[0].data_schema[i].name != col_name)
        {
            i++;
        }
        int acc_sums = child_results[0].data_schema[i].acc_col_size;
        char *data = call_sort_kernel(child_results[0].data.data(), row_size, child_results[0].num_row, acc_sums, (oredr_method == "ASC"));
        return_node_type return_data;
        return_data.data = std::vector<char>(data, data + child_results[0].num_row * row_size);
        return_data.num_row = child_results[0].num_row;
        return_data.data_schema = child_results[0].data_schema;
        return return_data;
    }
    else if (node->name == "AGGREGATE")
    {
        char *op;
        string col_name = node->details[0].substr(node->details[0].find("(") + 1, node->details[0].find(")") - node->details[0].find("(") - 1);
        int acc_col_size;
        int row_size = 0;
        int index;
        for (int i = 0; i < child_results[0].data_schema.size(); i++)
        {
            if (child_results[0].data_schema[i].name == col_name)
            {
                index = i;
                acc_col_size = child_results[0].data_schema[i].acc_col_size;
            }
            row_size += child_results[0].data_schema[i].size_in_bytes;
        }

        if (node->details[0].find("max") != std::string::npos)
        {
            op = "max";
        }
        else if (node->details[0].find("min") != std::string::npos)
        {
            op = "min";
        }
        else if (node->details[0].find("avg") != std::string::npos)
        {
            op = "avg";
        }
        else if (node->details[0].find("sum") != std::string::npos)
        {
            op = "sum";
        }
        else if (node->details[0].find("count") != std::string::npos)
        {
            op = "count";
        }

        double result = call_agg_kernel(child_results[0].data.data(), row_size, acc_col_size, op, child_results[0].num_row);
        const char *result_str = reinterpret_cast<const char *>(&result);
        return_node_type return_data;
        return_data.data = std::vector<char>(result_str, result_str + sizeof(result));
        return_data.num_row = 1;
        ColumnInfo col_info;
        col_info.type = child_results[0].data_schema[index].type;
        col_info.size_in_bytes = sizeof(result);
        col_info.acc_col_size = 0;
        col_info.name = node->details[0];
        return_data.data_schema.push_back(col_info);
        return return_data;
    }
    else if (node->name == "PROJECTION")
    {
        int *acc_sums = new int[node->details.size()];
        int *col_index = new int[node->details.size()];
        int *sizes = new int[node->details.size()];
        int new_row_size = 0;
        std::vector<ColumnInfo> child_schema = child_results[0].data_schema;
        std::vector<ColumnInfo> new_schema;
        int acc = 0;

        for (int i = 0; i < node->details.size(); i++)
        {
            std::string col_name = node->details[i];
            for (int j = 0; j < child_results[0].data_schema.size(); j++)
            {
                if (child_results[0].data_schema[j].name == col_name)
                {
                    ColumnInfo col_info = child_results[0].data_schema[j];
                    col_info.acc_col_size = acc;
                    acc += child_results[0].data_schema[j].size_in_bytes;
                    new_schema.push_back(col_info);
                    col_index[i] = j;
                    acc_sums[i] = child_results[0].data_schema[j].acc_col_size;
                    sizes[i] = child_results[0].data_schema[j].size_in_bytes;
                    new_row_size += child_results[0].data_schema[j].size_in_bytes;
                }
            }
        }

        int row_size = child_results[0].data.size() / child_results[0].num_row;

        std::cout << "New schema after projection:" << std::endl;
        char *data = call_project_kernel(child_results[0].data.data(), new_row_size, row_size, col_index, acc_sums, child_results[0].num_row, node->details.size(), sizes);
        return_node_type return_data;
        return_data.data = std::vector<char>(data, data + child_results[0].num_row * new_row_size);
        return_data.num_row = child_results[0].num_row;
        return_data.data_schema = new_schema;

        return return_data;
    }
    else
    {
        std::cout << "No matching kernel for: " << node->name << std::endl;
        return child_results[0]; // Return the first child's result as a fallback
    }
}

std::shared_ptr<PlanNode> build_plan_tree(LogicalOperator *op)
{
    if (!op)
        return nullptr;

    auto node = std::make_shared<PlanNode>();
    node->name = LogicalOperatorToString(op->type);

    // Handle LogicalGet
    if (auto get_op = dynamic_cast<duckdb::LogicalGet *>(op))
    {
        if (get_op->GetTable())
        {
            node->details.push_back(get_op->GetTable()->name);
        }
        for (auto &[col_idx, filter] : get_op->table_filters.filters)
        {
            std::ostringstream oss;
            oss << get_op->names[col_idx] << " ";
            switch (filter->filter_type)
            {
            case duckdb::TableFilterType::CONSTANT_COMPARISON:
            {
                auto &f = static_cast<duckdb::ConstantFilter &>(*filter);
                std::string constant_str = f.constant.ToString();
                if (f.constant.type().id() == duckdb::LogicalTypeId::VARCHAR &&
                    constant_str.front() != '\'' && constant_str.back() != '\'')
                {
                    constant_str = "'" + constant_str + "'";
                }
                oss << ExpressionTypeToString(f.comparison_type) << " " << constant_str;
                break;
            }
            case duckdb::TableFilterType::IS_NULL:
                oss << "IS NULL";
                break;
            case duckdb::TableFilterType::IS_NOT_NULL:
                oss << "IS NOT NULL";
                break;
            case duckdb::TableFilterType::CONJUNCTION_AND:
                oss << "AND";
                break;
            case duckdb::TableFilterType::CONJUNCTION_OR:
                oss << "OR";
                break;
            default:
                oss << "UNKNOWN";
                break;
            }
            node->details.push_back(oss.str());
        }
    }

    // Handle LogicalComparisonJoin
    if (op->type == LogicalOperatorType::LOGICAL_COMPARISON_JOIN)
    {
        auto &join_op = static_cast<duckdb::LogicalComparisonJoin &>(*op);
        for (auto &condition : join_op.conditions)
        {
            node->details.push_back("Join: " + condition.left->ToString() + " " + ExpressionTypeToString(condition.comparison) + " " + condition.right->ToString());
        }
    }

    // Handle LogicalOrder
    if (op->type == LogicalOperatorType::LOGICAL_ORDER_BY)
    {
        auto &order_by_op = static_cast<duckdb::LogicalOrder &>(*op);
        for (auto &order : order_by_op.orders)
        {
            std::string order_str = "Order By: " + order.expression->ToString();
            order_str += (order.type == duckdb::OrderType::ASCENDING) ? " ASC" : " DESC";
            node->details.push_back(order_str);
        }
    }

    for (auto &expr : op->expressions)
    {
        node->details.push_back(expr->ToString());
    }

    // Recurse on children
    for (auto &child : op->children)
    {
        node->children.push_back(build_plan_tree(child.get()));
    }

    return node;
}

void print_tree(std::shared_ptr<PlanNode> node, int indent = 0)
{
    if (!node)
        return;
    std::cout << std::string(indent, ' ') << "- " << node->name << std::endl;
    for (const auto &detail : node->details)
    {
        std::cout << std::string(indent + 2, ' ') << "* " << detail << std::endl;
    }
    for (const auto &child : node->children)
    {
        print_tree(child, indent + 4);
    }
}

std::unordered_map<std::string, std::string> remove_AS(string &query)
{
    std::unordered_map<std::string, std::string> alias_map;

    std::regex alias_pattern(R"(\b(\w+)\s+AS\s+(\w+))", std::regex::icase);
    std::smatch match;
    std::string::const_iterator searchStart(query.cbegin());

    while (std::regex_search(searchStart, query.cend(), match, alias_pattern))
    {
        std::string original = match[1];
        std::string alias = match[2];
        alias_map[original] = alias;
        searchStart = match.suffix().first;
    }

    for (const auto &pair : alias_map)
    {
        std::string pattern = "\\b" + pair.second + "\\b";
        query = std::regex_replace(query, std::regex(pattern), pair.first);
    }

    return alias_map;
}

int main(int argc, char *argv[])
{
    // DuckDB
    using namespace duckdb;
    DuckDB db(nullptr);
    Connection con(db);
    ClientContext &context = *con.context;
    con.Query("SET disabled_optimizers='filter_pushdown,statistics_propagation';");

    while (true)
    {
        cout << "\nEnter SQL query (or type 'exit' to quit): ";
        string query;
        getline(cin, query);

        if (query == "exit" || query == "quit")
        {
            cout << "Exiting CLI.\n";
            break;
        }

        std::unordered_map<std::string, std::string> alias_map = remove_AS(query);
        auto start_time = std::chrono::high_resolution_clock::now();
        get_schema(schema);
        create_tables_from_schema(con, schema);

        Parser parser;
        parser.ParseQuery(query);
        auto statements = std::move(parser.statements);
        // Start a transaction
        con.BeginTransaction(); // Start transaction using Connection

        // Create a planner and plan the query
        Planner planner(context);
        planner.CreatePlan(std::move(statements[0]));

        // Now you can proceed with further processing or optimization
        cout << "Planning successful!" << endl;
        cout << "Unoptimized Logical Plan:\n"
             << planner.plan->ToString() << endl;

        Optimizer optimizer(*planner.binder, context);
        auto logical_plan = optimizer.Optimize(std::move(planner.plan));
        cout << "Optimized Logical Plan:\n";
        cout << logical_plan->ToString() << endl;

        auto tree_root = build_plan_tree(logical_plan.get());
        print_tree(tree_root);

        return_node_type data_out = post_order_traverse_and_launch_kernel(tree_root);

        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_time = end_time - start_time;
        std::cout << "Query execution time on GPU : " << elapsed_time.count() << " seconds" << std::endl;
        print_chunk(data_out.data, data_out.data_schema, alias_map);

        // auto start_time = std::chrono::high_resolution_clock::now();
        // get_schema(schema);
        // create_tables_from_schema(con, schema);
        // con.BeginTransaction();
        // con.Query(query);
        // auto end_time = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> elapsed_time = end_time - start_time;
        // std::cout << "Query execution time on CPU : " << elapsed_time.count() << " seconds" << std::endl;
        con.Commit(); // Commit transaction using Connection
    }
}
